#include "hip/hip_runtime.h"
#include "fix_gpu.cuh"
#include "image.hh"
#include "map_fix_pixels.cuh"

#include <array>
#include <numeric>
#include <algorithm>
#include <cmath>

void fix_image_gpu(Image& to_fix, const raft::handle_t handle)
{
    const int image_size = to_fix.width * to_fix.height;

    // #1 Compact

    // Build predicate vector

    std::vector<int> predicate(to_fix.size(), 0);

    constexpr int garbage_val = -27;
    for (int i = 0; i < to_fix.size(); ++i)
        if (to_fix.buffer[i] != garbage_val)
            predicate[i] = 1;

    // Compute the exclusive sum of the predicate

    std::exclusive_scan(predicate.begin(), predicate.end(), predicate.begin(), 0);

    // Scatter to the corresponding addresses

    for (std::size_t i = 0; i < predicate.size(); ++i)
        if (to_fix.buffer[i] != garbage_val)
            to_fix.buffer[predicate[i]] = to_fix.buffer[i];


    // #2 Apply map to fix pixels

    map_fix_pixels(raft::device_span<int>(to_fix.buffer, image_size),
                    handle.get_stream());

    // #3 Histogram equalization

    // Histogram

    std::array<int, 256> histo;
    histo.fill(0);
    for (int i = 0; i < image_size; ++i)
        ++histo[to_fix.buffer[i]];

    // Compute the inclusive sum scan of the histogram

    std::inclusive_scan(histo.begin(), histo.end(), histo.begin());

    // Find the first non-zero value in the cumulative histogram

    auto first_none_zero = std::find_if(histo.begin(), histo.end(), [](auto v) { return v != 0; });

    const int cdf_min = *first_none_zero;

    // Apply the map transformation of the histogram equalization

    std::transform(to_fix.buffer, to_fix.buffer + image_size, to_fix.buffer,
        [image_size, cdf_min, &histo](int pixel)
            {
                return std::roundf(((histo[pixel] - cdf_min) / static_cast<float>(image_size - cdf_min)) * 255.0f);
            }
    );
}