#include "hip/hip_runtime.h"
#include "scatter.cuh"

__global__
void scatter_kernel(raft::device_span<int> buffer, raft::device_span<int> predicate, raft::device_span<int> res_image_buffer, const int garbage_val) 
{
    unsigned int id = threadIdx.x + blockIdx.x * blockDim.x;

    if (id >= buffer.size())
        return;

    int val = buffer[id];
    if (val != garbage_val)
        res_image_buffer[predicate[id]] = val;
}

void scatter(raft::device_span<int> buffer, raft::device_span<int> predicate, raft::device_span<int> res_image_buffer, const int garbage_val, hipStream_t stream) 
{
    int block_size = 512;
    int grid_size = (buffer.size() + block_size - 1) / block_size;
    scatter_kernel<<<grid_size, block_size, 0, stream>>>(buffer, predicate, res_image_buffer, garbage_val);
    CUDA_CHECK_ERROR(hipGetLastError());
}