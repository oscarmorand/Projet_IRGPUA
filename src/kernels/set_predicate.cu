#include "hip/hip_runtime.h"
#include "set_predicate.cuh"

__global__
void set_predicate_kernel(raft::device_span<int> buffer, raft::device_span<int> predicate, const int garbage_val, const int padded_size) 
{
    unsigned int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id >= padded_size)
        return;
    if (id >= buffer.size()) {
        predicate[id] = 0;
        return;
    }
    predicate[id] = (buffer[id] != garbage_val);
}

void set_predicate(raft::device_span<int> to_fix, raft::device_span<int> predicate, const int garbage_val, 
                   const int padded_size, hipStream_t stream)
{
    const int block_size = 1024;
    int gridsize = (to_fix.size() + block_size - 1) / block_size;
    set_predicate_kernel<<<gridsize, block_size, 0, stream>>>(to_fix, predicate, garbage_val, padded_size);
    CUDA_CHECK_ERROR(hipGetLastError());
}