#include "hip/hip_runtime.h"
#include "scatter.cuh"

__global__
void scatter_kernel(raft::device_span<int> buffer, raft::device_span<int> predicate, const int garbage_val) 
{
    unsigned int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i >= buffer.size())
        return;
    if (buffer[i] != garbage_val)
        buffer[predicate[i]] = buffer[i];
}

void scatter(raft::device_span<int> buffer, raft::device_span<int> predicate, const int garbage_val, hipStream_t stream) 
{
    int block_size = 1024;
    int grid_size = (buffer.size() + block_size - 1) / block_size;
    scatter_kernel<<<grid_size, block_size, 0, stream>>>(buffer, predicate, garbage_val);
    CUDA_CHECK_ERROR(hipGetLastError());
}